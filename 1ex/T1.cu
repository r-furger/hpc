
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define DATASIZE 64 
__global__ void get_average(double *in, double *out, int width)
{
        int id=threadIdx.x;
        if(id==0||id==width-1)
        {
                out[id]=in[id];
        }
        else
        {
                out[id]=(in[id-1]+in[id]+in[id+1])/3;
        }
}
void init( double * input, int width )
{
        int i;
        for(i=0;i<width;i++)
        {
                input[i]= rand() % 5;
        }
}
int check(double * original, double * averaged, int width)
{
        int i;
        for(i=1;i<width-1;i++)
        {
                if(averaged[i]!=(original[i-1]+original[i]+original[i+1])/3)
                {
                        printf("Something goes wrong :(\n");
                        return -1;
                }
        }
        if(averaged[0]!=original[0]||averaged[width-1]!=original[width-1])
        {
                printf("Boundary condition are not fine :( \n");
                        return -1;
        }
        printf("Correct solution\n");
        return 0;
}
int main()
{
        double * d_in;
        double * d_out;
        // Memory allocation at host sid
        double * h_in = (double *)malloc(sizeof(double)*DATASIZE);
        double * h_out= (double *)malloc(sizeof(double)*DATASIZE);
        // CHANGE: Memory allocation at device side
        hipMalloc(&d_in,sizeof(double)* DATASIZE);
        hipMalloc(&d_out,sizeof(double)* DATASIZE);
        
        // create random 1d array
        init(h_in,DATASIZE);
        // copy from host to device (GPU) init data
        hipMemcpy(d_in,h_in,DATASIZE,hipMemcpyHostToDevice);
        // kernel
        get_average<<<1,DATASIZE>>>(d_in,d_out,DATASIZE*sizeof(double));
        // CHANGE: copy resulting data from device to host
        //cudaMemcpy(h_out,d_out,sizeof(double)*DATASIZE,cudaMemcpyHostToDevice);
        hipMemcpy(h_out,d_out,sizeof(double)*DATASIZE,hipMemcpyDeviceToHost);
        
        check(h_in,h_out,DATASIZE);
        
        // Free device, host memory
        hipFree(d_in);
        // CHANGE:
        hipFree(d_out);
        free(h_in);
        free(h_out);
}