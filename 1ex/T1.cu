
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define DATASIZE 64 
__global__ void get_average(double *in, double *out, int width)
{
        int id=threadIdx.x;
        if(id==0||id==width-1)
        {
                out[id]=in[id];
        }
        else
        {
                out[id]=(in[id-1]+in[id]+in[id+1])/3;
        }
}
void init( double * input, int width )
{
        int i;
        for(i=0;i<width;i++)
        {
                input[i]= rand() % 5;
        }
}
int check(double * original, double * averaged, int width)
{
        int i;
        for(i=1;i<width-1;i++)
        {
                if(averaged[i]!=(original[i-1]+original[i]+original[i+1])/3)
                {
                        printf("Something goes wrong :(\n");
                        return -1;
                }
        }
        if(averaged[0]!=original[0]||averaged[width-1]!=original[width-1])
        {
                printf("Boundary condition are not fine :( \n");
                        return -1;
        }
        printf("Correct solution\n");
        return 0;
}
int main()
{
        double * d_in;
        double * d_out;
        double * h_in = (double *)malloc(sizeof(double)*DATASIZE);
        double * h_out= (double *)malloc(sizeof(double)*DATASIZE);
        init(h_in,DATASIZE);
        hipMemcpy(d_in,h_in,DATASIZE,hipMemcpyHostToDevice);
        get_average<<<1,DATASIZE>>>(d_in,d_out,DATASIZE*sizeof(double));
        hipMemcpy(h_out,d_out,sizeof(double)*DATASIZE,hipMemcpyHostToDevice);
        check(h_in,h_out,DATASIZE);
        hipFree(d_in);
        free(h_in);
        free(h_out);
}