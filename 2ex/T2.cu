#include "hip/hip_runtime.h"
#define N 1024 // number of rows = number of columns
#include <stdio.h>
__global__ void matrix_mult_kernel (int *a, int *b, int *c, int width);
void init(int * input,int length);
void print_matrix(int * matrix, int size);
int main()
{
 	int * h_a,*d_a;
 	int * h_b,*d_b;
 	int * h_c,*d_c;
 	int data_length = N * N * sizeof(int);
 	h_a=(int*)malloc(data_length);
 	h_b=(int*)malloc(data_length);
 	h_c=(int*)malloc(data_length);
 
 	init(h_a,N*N);
 	init(h_b,N*N);

 	// TODO: Allocate the matrix d_a, d_b, and d_c at the device memory
 	hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
    printf("cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
    }
 	
    // TODO: Copy input matrix h_a and h_b to the device memory
    err = hipGetLastError();
 	if ( hipSuccess != err )
    {
    printf("cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
    }
 	
    // TODO: Launch your matrix multiplication kernel
 	err = hipGetLastError();
    if ( hipSuccess != err )
    {
    printf("cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
    }

 	// TODO: Copy output matrix h_c to the host memory 
 	err = hipGetLastError();
    if ( hipSuccess != err )
    {
    printf("cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
    }
 	
    // TODO: Free all alocated memory
 	err = hipGetLastError();
    if ( hipSuccess != err )
    {
    printf("cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
    }
 	
    free(h_a);
 	free(h_b);
 	free(h_c);
}
void init(int * input, int size)
{
	int i;
	for(i=0;i<size;i++)
	{
		input[i]=rand()%5;
	}
}
void print_matrix(int * matrix,int size)
{
	printf("Matrix items: \n");
	int i,j;
	for(i=0;i<size;i++)
	{
		for(j=0;j<size;j++)
			printf("%d,",matrix[i*size+j]);
		printf("\n");
	}
}
__global__ void matrix_mult_kernel (int *a, int *b, int *c, int width)
 {
    //TODO implement the naive matrix multiplication version
 }
}
